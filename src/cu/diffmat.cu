#include "hip/hip_runtime.h"
/**
 * @file diffmat.cu
 * @author Daniel San Martin (dsanmartinreyes@gmail.com)
 * @brief Build differentiation matrices in device
 * @version 0.1
 * @date 2020-09-01
 * 
 * @copyright Copyright (c) 2020
 * 
 */

#include <stdlib.h>
#include "include/diffmat.cuh"

/**
 * @brief Finite difference matrix for first derivative 
 * 
 * @param D1N Pointer to fill with the matrix
 * @param M Size of rows/columns of array
 * @param h \f$ \Delta x \f$ or \f$ \Delta y \f$
 */
__global__ void FD1Kernel(double *D1N, int M, double h) {
    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    if (tId < M * M) {
        int i = tId % M; // Row index
        int j = tId / M; // Col index
        if (i - j == -1) 
            D1N[j * M + i] = 1 / (2 * h);
        else if (i - j == 1)
            D1N[j * M + i] = -1 / (2 * h);
            
        if (i == 0)
            D1N[M * (M - 1)] = -1 / (2 * h);
        if (i == M - 1)
            D1N[M - 1] = 1 / (2 * h);
    }
}

/**
 * @brief Finite difference matrix for second derivative
 * 
 * @param D2N Pointer for the matrix
 * @param M Number of rows/cols
 * @param h \f$ Delta x \f$ or \f$ \Delta y \f$
 */
__global__ void FD2Kernel(double *D2N, int M, double h) {
    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    if (tId < M * M) {
        int i = tId % M; // Row index
        int j = tId / M; // Col index
        if (i == j) {
            D2N[j * M + i] = -2 / (h * h);
        } else if (abs(i - j) == 1) {
            D2N[j * M + i] = 1 / (h * h);
        }
        if (i == 0)
            D2N[M * (M - 1)] = 1 / (h * h);
        if (i == M - 1)
            D2N[M - 1] = 1 / (h * h);
    }
}

/**
 * @brief Fill array with Chebyshev nodes
 * 
 * @param x_c Pointer array
 * @param N Number of nodes
 */
__global__ void ChebyshevNodes(double *x_c, int N) {
    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    if (tId < N + 1) {
        x_c[tId] = cos(tId * M_PI / N);
    }
}

/**
 * @brief Build first derivative Chebyshev matrix
 * 
 * @param CDM Pointer array to fill
 * @param x_c Chebyshev nodes pointer
 * @param N Number of nodes
 */
__global__ void ChebyshevMatrix(double *CDM, double *x_c, int N) {
    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    if (tId < (N + 1) * (N + 1)) {
        double c_i, c_j;
        int i = tId % (N + 1); // Row index
        int j = tId / (N + 1); // Col index
        if (i == 0 && j == 0) {
            CDM[j * (N + 1) + i] = (2.0 * N * N + 1.0) / 6.0;
        } else if (i == N && j == N) {
            CDM[j * (N + 1) + i] = - (2.0 * N * N + 1.0) / 6.0;
        } else if (i == j) {
            CDM[j * (N + 1) + i] = - x_c[j] / (2.0 * (1.0 - x_c[j] * x_c[j]));
        } else {
            c_i = (i == 0 || i == N) ? 2.0 : 1.0;
            c_j = (j == 0 || j == N) ? 2.0 : 1.0;
            CDM[j * (N + 1) + i] = c_i * pow(-1.0, i + j) / (c_j * (x_c[i] - x_c[j]));
        }
    }
}

/**
 * @brief Compute second derivative Chebyshev differentiation matrix
 * 
 * @param CDM2 Pointer to fill with matrix
 * @param CDM First derivative Chebyshev differentiation matrix
 * @param N Number of nodes
 */
__global__ void Chebyshev2Matrix(double *CDM2, double *CDM, int N) {
    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    if (tId < (N + 1) * (N + 1)) {
        int i = tId % (N + 1); // Row index
        int j = tId / (N + 1); // Col index
        double c = 0;
        for (int k = 0; k < N + 1; k++) {
            c += CDM[k * (N + 1) + i] * CDM[j * (N + 1) + k];
        }
        CDM2[tId] = c;
    }
}