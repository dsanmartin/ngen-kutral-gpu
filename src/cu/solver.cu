#include <stdio.h>
#include "include/solver.cuh"
#include "include/diffmat.cuh"
#include "../c/include/files.h"

void RHS(double *Unew, double *Uold, double *V1, double *V2, double *Dx, double *Dy, double *Dxx, double *Dyy,
	double *tmp, double kappa, double dt, int Nx, int Ny) {

	int lda=Ny, ldb=Nx, ldc=Ny;
	const double alf = kappa * dt;
	const double bet = 1;
	const double *alpha = &alf;
	const double *beta = &bet;
	const double v1 = -0.707107 * dt;
	const double v2 = -0.707107 * dt; 
	const double *av1 = &v1;
	const double *av2 = &v2;

	// Create handles for CUBLAS
	hipblasHandle_t handle, handle2, handle3, handle4;
	hipblasCreate(&handle);
	hipblasCreate(&handle2);
	hipblasCreate(&handle3);
	hipblasCreate(&handle4);

	// For euler method, copy u_old to u_new
	hipMemcpy(Unew, Uold, Nx * Ny * sizeof(double), hipMemcpyDeviceToDevice);

	/* Compute Diffusion */
	// Compute: kappa*dt D_yy U_old + "U_old"
	hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, Ny, Nx, Nx, alpha, Dyy, lda, Uold, ldb, beta, Unew, ldc);
	
	// Wait first matrix multiplication to reuse it
	hipDeviceSynchronize();

	// Compute: kappa*dt U_old D_xx^T + kappa*dt D_yy U_old + "U_old"
	hipblasDgemm(handle2, HIPBLAS_OP_N, HIPBLAS_OP_T, Ny, Nx, Nx, alpha, Uold, lda, Dxx, ldb, beta, Unew, ldc);
	/* End Diffusion computation */

	// Wait diffusion computation
	hipDeviceSynchronize();

	/* Compute Convection*/
	// Compute: -v2 * dt D_y U_old + diffusion
	hipblasDgemm(handle3, HIPBLAS_OP_N, HIPBLAS_OP_N, Ny, Nx, Nx, av2, Dy, lda, Uold, ldb, beta, Unew, ldc);

	// Wait for convection in y
	hipDeviceSynchronize();

	// Compute: -v1 * dt U_old * D_x^T + (diffusion - y convection)
	hipblasDgemm(handle4, HIPBLAS_OP_N, HIPBLAS_OP_T, Ny, Nx, Nx, av1, Uold, lda, Dx, ldb, beta, Unew, ldc);
	/* End convection computation */

	// Destroy the handles
	hipblasDestroy(handle);
	hipblasDestroy(handle2);
	hipblasDestroy(handle3);
	hipblasDestroy(handle4);
}


void solver(double *h_U0, double *h_V1, double *h_V2, double *h_U, int Nx, int Ny, int T, 
	double dx, double dy, double dt, double kappa) {
	double *d_U, *d_V1, *d_V2, *d_Dx, *d_Dy, *d_Dxx, *d_Dyy, *d_tmp;

	/* Create differentiation matrices for second derivative */
	double *h_Dx = (double *)malloc(Nx * Nx * sizeof(double));
	double *h_Dy = (double *)malloc(Ny * Ny * sizeof(double));
	double *h_Dxx = (double *)malloc(Nx * Nx * sizeof(double));
	double *h_Dyy = (double *)malloc(Ny * Ny * sizeof(double));
	double *h_tmp = (double *)malloc(Ny * Nx * sizeof(double)); // Temporal matrix for computations
	FD1(h_Dx, Nx, dx); // Fill differentiation matrix without boundaries
	FD1(h_Dy, Ny, dy); // Fill differentiation matrix without boundaries
	FD2(h_Dxx, Nx, dx); // Fill second differentiation matrix without boundaries
	FD2(h_Dyy, Ny, dy); // Fill second differentiation matrix without boundaries

	/* Copy initial condition to temperatures approximation */
	memcpy(h_U, h_U0, (Nx * Ny) * sizeof(double));

	/* Memory allocation for matrices in GPU */
	hipMalloc(&d_U, T * Ny * Nx * sizeof(double));
	hipMalloc(&d_Dx, Nx * Nx * sizeof(double));
	hipMalloc(&d_Dy, Ny * Ny * sizeof(double));
	hipMalloc(&d_Dxx, Nx * Nx * sizeof(double));
	hipMalloc(&d_Dyy, Ny * Ny * sizeof(double));
	hipMalloc(&d_V1, Ny * Nx * sizeof(double));
	hipMalloc(&d_V2, Ny * Nx * sizeof(double));
	hipMalloc(&d_tmp, Ny * Nx * sizeof(double));

	/* Copy to GPU */
	hipMemcpy(d_U, h_U, Ny * Nx * T * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_Dx, h_Dx, Nx * Nx * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_Dy, h_Dy, Ny * Ny * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_Dxx, h_Dxx, Nx * Nx * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_Dyy, h_Dyy, Ny * Ny * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_V1, h_V1, Ny * Nx * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_V2, h_V2, Ny * Nx * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_tmp, h_tmp, Ny * Nx * sizeof(double), hipMemcpyHostToDevice);

	/* ODE Solver 	*/
	for (int t = 1; t < T; t++) {
		//printf("t: %d", t * Nx * Ny);
		//printMatrix(&d_U[t * Nx * Ny], Ny, Nx);
		//RHS(&d_U[t * (Nx-1) * (Ny-1)], &d_U[(t - 1) * (Nx-1) * (Ny-1)], d_V1, d_V2, d_Dxx, d_Dyy, kappa, dt, Nx-2, Ny-2);
		RHS(&d_U[t * Nx * Ny], &d_U[(t - 1) * Nx * Ny], d_V1, d_V2, d_Dx, d_Dy, d_Dxx, d_Dyy, d_tmp, kappa, dt, Nx, Ny);
	}
	
	// Copy from device to host
	hipMemcpy(h_U, d_U, T * Ny * Nx * sizeof(double), hipMemcpyDeviceToHost);

	// Free device memory
	hipFree(d_U);
	hipFree(d_V1);
	hipFree(d_V2);
	hipFree(d_Dx);
	hipFree(d_Dy);
	hipFree(d_Dxx);
	hipFree(d_Dyy);
	hipFree(d_tmp);

	// Free host memory
	free(h_Dx);
	free(h_Dy);
	free(h_Dxx);
	free(h_Dyy);
	free(h_tmp);
}