#include "hip/hip_runtime.h"
#include <stdio.h>
#include "include/wildfire.cuh"
#include "include/diffmat.cuh"
#include "include/utils.cuh"
#include "../c/include/files.h"
#include "../c/include/utils.h"

#define DB 256 // Threads per block
#define DG(size) (size + DB - 1) / DB // Blocks per grid

__constant__ double buffer[256];

/* Gaussian kernel */
__device__ double gaussian(double A, double sigma_x, double sigma_y, double x, double y) {
	return A * exp((x * x) / sigma_x + (y * y) / sigma_y);
}

/* Temperature initial condition */
__device__ double u0(double x, double y) {
	return gaussian(6.0, -20.0, -20.0, x, y);
}

/* Fuel initial condition */
__device__ double b0(double x, double y) {
	return 1;
}

__device__ double v1(double x, double y) {
	return 0.70710678118;
}

__device__ double v2(double x, double y) {
	return 0.70710678118;
}

__device__ double s(double u, double upc) {
	return u >= upc;
}

__device__ double f(Parameters parameters, double u, double b) {
	return s(u, parameters.upc) * b * exp(u / (1 + parameters.epsilon * u)) - parameters.alpha * u;
}

__device__ double g(Parameters parameters, double u, double b) {
	return -s(u, parameters.upc) * (parameters.epsilon / parameters.q) * b * exp(u /(1 + parameters.epsilon * u));
}

__global__ void U0(Parameters parameters, double *U, double x_ign, double y_ign) {
	int tId = threadIdx.x + blockIdx.x * blockDim.x;
	if (tId < parameters.N * parameters.M) {
		int i = tId % parameters.M; // Row index
		int j = tId / parameters.M; // Col index
		double u = 0; // Boundary condition
		if (!(i == 0 || i == parameters.M - 1 || j == 0 || j == parameters.N - 1))
			u = u0(buffer[j] - x_ign, buffer[parameters.N + i] - y_ign);
		U[j * parameters.M + i] = u;
	}
}

__global__ void B0(Parameters parameters, double *B) {
	int tId = threadIdx.x + blockIdx.x * blockDim.x;
	if (tId < parameters.N * parameters.M) {
		int i = tId % parameters.M; // Row index
		int j = tId / parameters.M; // Col index
		double b = 0;
		if (!(i == 0 || i == parameters.M - 1 || j == 0 || j == parameters.N - 1))
			b = b0(buffer[j], buffer[parameters.N + i]);
		B[j * parameters.M + i] = b;
	}
}

__device__ double RHSU(Parameters parameters, DiffMats DM, double *Y, int i, int j) {
	/* Get actual value of approximations */
	double u = Y[j * parameters.M + i];
	double b = Y[j * parameters.M + i + parameters.M * parameters.N];

	/* Evaluate vector field */
	double v_v1 = v1(buffer[j], buffer[parameters.N + i]);
	double v_v2 = v2(buffer[j], buffer[parameters.N + i]);  
	
	/* Compute derivatives */
	double ux = 0.0, uy = 0.0, uxx = 0.0, uyy = 0.0;
	int m = parameters.M;
	int n = parameters.N;
	for (int k = 0; k < parameters.N; k++) {
		ux += Y[k * m + i] * DM.Dx[k * n + j];
		uy += DM.Dy[k * m + i] * Y[j * m + k];
		uxx += Y[k * m + i] * DM.Dxx[k * n + j];
		uyy += DM.Dyy[k * m + i] * Y[j * m + k];
	}

	/* Compute PDE */
	double diffusion = parameters.kappa * (uxx + uyy);
	double convection = v_v1 * ux + v_v2 * uy;
	double reaction = f(parameters, u, b);
	return diffusion - convection + reaction;
}

__device__ double RHSB(Parameters parameters, double *Y, int i, int j) {
	double u = Y[j * parameters.M + i];
	double b = Y[j * parameters.M + i + parameters.M * parameters.N];
	return g(parameters, u, b);
}

__global__ void simulationBlock(Parameters parameters, DiffMats DM, double *Y, double *Y_old, double dt) {
	int sim = blockIdx.x;
	int index = threadIdx.x;
	int offset = 2 * sim  * parameters.M * parameters.N;
	for (int k = 1; k <= parameters.L; k++) { 

		for (int nodes = 0; nodes <= 2 * parameters.M * parameters.N; nodes++) 
			Y_old[nodes] = Y[offset + nodes];

		while (index < parameters.M * parameters.N) {
			int i = index % parameters.M; // Row index
			int j = index / parameters.M; // Col index
			double u_new = 0; // Boundary conditions
			double b_new = 0; // Boundary conditions
			
			int gindex = offset + j * parameters.M + i;

			/* Get actual value of approximations */
			double u_old = Y_old[gindex];
			double b_old = Y_old[gindex + parameters.M * parameters.N];

			/* PDE */
			if (!(i == 0 || i == parameters.M - 1 || j == 0 || j == parameters.N - 1)) { // Inside domain
				double fuel = g(parameters, u_old, b_old);
				u_new = RHSU(parameters, DM, Y_old + offset, i, j);
				b_new = fuel;
			}
			/* Update values using Euler method */
			Y[gindex] = u_old + dt * u_new;
			Y[gindex + parameters.M * parameters.N] = b_old + dt * b_new;
			index += blockDim.x;
		}
		__syncthreads();
	}
}

/*
	Right hand side using Euler method.
	This approach use all threads to compute each node of all simulations.
	Kernel time: 67.005ms 
*/
__global__ void RHSEuler(Parameters parameters, DiffMats DM, double *Y, double *Y_old, double dt) {
	int tId = threadIdx.x + blockIdx.x * blockDim.x;
	int n_sim = parameters.x_ign_n * parameters.y_ign_n;
  if (tId < n_sim * parameters.M * parameters.N) {
		int sim = tId / (parameters.M * parameters.N);
		int i = (tId - sim * parameters.M * parameters.N) % parameters.M; // Row index
		int j = (tId - sim * parameters.M * parameters.N) / parameters.M; // Col index
    double u_new = 0; // Boundary conditions
		double b_new = 0; // Boundary conditions
		int offset = 2 * sim  * parameters.M * parameters.N;
		int gindex = offset + j * parameters.M + i;

		/* Get actual value of approximations */
		double u_old = Y_old[gindex];
		double b_old = Y_old[gindex + parameters.M * parameters.N];

		/* PDE */
    if (!(i == 0 || i == parameters.M - 1 || j == 0 || j == parameters.N - 1)) { // Inside domain
      double fuel = g(parameters, u_old, b_old);
      u_new = RHSU(parameters, DM, Y_old + offset, i, j);
      b_new = fuel;
		}

		/* Update values using Euler method */
    Y[gindex] = u_old + dt * u_new;
		Y[gindex + parameters.M * parameters.N] = b_old + dt * b_new;
  }
}

/*
	Right hand side using Euler method.
	This approach use each block for a single simulation.
	Kernel time: 54.407ms 
*/
__global__ void RHSEulerBlock(Parameters parameters, DiffMats DM, double *Y, double *Y_old, double dt) {
	int sim = blockIdx.x;
	int index = threadIdx.x;
  while (index < parameters.M * parameters.N) {
		int i = index % parameters.M; // Row index
		int j = index / parameters.M; // Col index
    double u_new = 0; // Boundary conditions
		double b_new = 0; // Boundary conditions
		int offset = 2 * sim  * parameters.M * parameters.N;
		int gindex = offset + j * parameters.M + i;

		/* Get actual value of approximations */
		double u_old = Y_old[gindex];
		double b_old = Y_old[gindex + parameters.M * parameters.N];

		/* PDE */
    if (!(i == 0 || i == parameters.M - 1 || j == 0 || j == parameters.N - 1)) { // Inside domain
      double fuel = g(parameters, u_old, b_old);
      u_new = RHSU(parameters, DM, Y_old + offset, i, j);
      b_new = fuel;
		}
		/* Update values using Euler method */
    Y[gindex] = u_old + dt * u_new;
		Y[gindex + parameters.M * parameters.N] = b_old + dt * b_new;
		index += blockDim.x;
	}
}

__global__ void sumVector(Parameters parameters, double *c, double *a, double *b, double scalar, int size) {
	int tId = threadIdx.x + blockIdx.x * blockDim.x;
	if (tId < size) {
		c[tId] = a[tId] + scalar * b[tId];
	}
}

/* Compute RHS using all threads 43.5ms */
__global__ void RHSvec(Parameters parameters, DiffMats DM, double *k, double *vec) {
	int tId = threadIdx.x + blockIdx.x * blockDim.x;
	int n_sim = parameters.x_ign_n * parameters.y_ign_n;
	if (tId < n_sim * parameters.M * parameters.N) {
		int sim = tId / (parameters.M * parameters.N);
		int i = (tId - sim * parameters.M * parameters.N) % parameters.M; // Row index
		int j = (tId - sim * parameters.M * parameters.N) / parameters.M; // Col index
		int offset = 2 * sim  * parameters.M * parameters.N;
		int gindex = offset + j * parameters.M + i;

		int u_index = gindex;//j * parameters.M + i;
		int b_index = parameters.M * parameters.N + u_index;
		double u_k = 0;
		double b_k = 0;
		if (!(i == 0 || i == parameters.M - 1 || j == 0 || j == parameters.N - 1)) {
			u_k = RHSU(parameters, DM, vec + offset, i, j);
			b_k = g(parameters, vec[u_index], vec[b_index]);
		}
		k[u_index] = u_k;
		k[b_index] = b_k;
	}
}

/* Compute RHS using a block per simulation */
__global__ void RHSvecBlock(Parameters parameters, DiffMats DM, double *k, double *vec) {
	int sim = blockIdx.x;
	int index = threadIdx.x;
	while (index < parameters.M * parameters.N) {
		int i = index % parameters.M; // Row index
		int j = index / parameters.M; // Col index
		int offset = 2 * sim  * parameters.M * parameters.N;
		int gindex = offset + j * parameters.M + i;

		int u_index = gindex;
		int b_index = parameters.M * parameters.N + u_index;
		double u_k = 0;
		double b_k = 0;
		if (!(i == 0 || i == parameters.M - 1 || j == 0 || j == parameters.N - 1)) {
			u_k = RHSU(parameters, DM, vec + offset, i, j);
			b_k = g(parameters, vec[u_index], vec[b_index]);
		}
		k[u_index] = u_k;
		k[b_index] = b_k;
		index += blockDim.x;
	}
}

/*
	Right hand side using RK4 method.
	This approach use all threads to compute each node of all simulations.
	Kernel time: 1.8705ms 
*/
__global__ void RHSRK4(Parameters parameters, DiffMats DM, double *Y, double *Y_old, 
	double *k1, double *k2, double *k3, double *k4, double dt) {
	int tId = threadIdx.x + blockIdx.x * blockDim.x;
	int n_sim = parameters.x_ign_n * parameters.y_ign_n;
  if (tId < n_sim * parameters.M * parameters.N) {
		int sim = tId / (parameters.M * parameters.N);
		int i = (tId - sim * parameters.M * parameters.N) % parameters.M; // Row index
		int j = (tId - sim * parameters.M * parameters.N) / parameters.M; // Col index
    double u_new = 0; // Boundary conditions
		double b_new = 0; // Boundary conditions
		int offset = 2 * sim  * parameters.M * parameters.N;
		int gindex = offset + j * parameters.M + i;

		/* Get actual value of approximations */
		double u_old = Y_old[gindex];
		double b_old = Y_old[gindex + parameters.M * parameters.N];
		double u_k1 = 0, u_k2 = 0, u_k3 = 0, u_k4 = 0;
		double b_k1 = 0, b_k2 = 0, b_k3 = 0, b_k4 = 0; 

		/* PDE */
    if (!(i == 0 || i == parameters.M - 1 || j == 0 || j == parameters.N - 1)) { // Inside domain
			u_k1 = k1[gindex];
			u_k2 = k2[gindex];
			u_k3 = k3[gindex];
			u_k4 = k4[gindex];
			b_k1 = k1[gindex + parameters.M * parameters.N];
			b_k2 = k2[gindex + parameters.M * parameters.N];
			b_k3 = k3[gindex + parameters.M * parameters.N];
			b_k4 = k4[gindex + parameters.M * parameters.N];
			u_new = u_k1 + 2 * u_k2 + 2 * u_k3 + u_k4;
			b_new = b_k1 + 2 * b_k2 + 2 * b_k3 + b_k4;
		}

		/* Update values using RK4 method */
    Y[gindex] = u_old + (1.0 / 6.0) * dt * u_new;
		Y[gindex + parameters.M * parameters.N] = b_old + (1.0 / 6.0) * dt * b_new;
  }
}

/*
	Right hand side using RK4 method.
	This approach use each block for a single simulation.
	Kernel time: 2.642ms
*/
__global__ void RHSRK4Block(Parameters parameters, DiffMats DM, double *Y, double *Y_old,
	double *k1, double *k2, double *k3, double *k4, double dt) {
	int sim = blockIdx.x;
	int index = threadIdx.x;
  while (index < parameters.M * parameters.N) {
		int i = index % parameters.M; // Row index
		int j = index / parameters.M; // Col index
    double u_new = 0; // Boundary conditions
		double b_new = 0; // Boundary conditions
		int offset = 2 * sim  * parameters.M * parameters.N;
		int gindex = offset + j * parameters.M + i;

		/* Get actual value of approximations */
		double u_old = Y_old[gindex];
		double b_old = Y_old[gindex + parameters.M * parameters.N];
		double u_k1 = 0, u_k2 = 0, u_k3 = 0, u_k4 = 0;
		double b_k1 = 0, b_k2 = 0, b_k3 = 0, b_k4 = 0; 

		/* PDE */
    if (!(i == 0 || i == parameters.M - 1 || j == 0 || j == parameters.N - 1)) { // Inside domain
      u_k1 = k1[gindex];
			u_k2 = k2[gindex];
			u_k3 = k3[gindex];
			u_k4 = k4[gindex];
			b_k1 = k1[gindex + parameters.M * parameters.N];
			b_k2 = k2[gindex + parameters.M * parameters.N];
			b_k3 = k3[gindex + parameters.M * parameters.N];
			b_k4 = k4[gindex + parameters.M * parameters.N];
			u_new = u_k1 + 2 * u_k2 + 2 * u_k3 + u_k4;
			b_new = b_k1 + 2 * b_k2 + 2 * b_k3 + b_k4;
		}

		/* Update values using RK4 method */
    Y[gindex] = u_old + (1.0 / 6.0) * dt * u_new;
		Y[gindex + parameters.M * parameters.N] = b_old + (1.0 / 6.0) * dt * b_new;
		index += blockDim.x;
	}
	__syncthreads();
}

void ODESolver(Parameters parameters, DiffMats DM, double *d_Y, double dt) {
	int n_sim = parameters.x_ign_n * parameters.y_ign_n; // Number of wildfire simulations
	int size = 2 * n_sim * parameters.M * parameters.N;

	/* Time method */
	if (strcmp(parameters.time, "Euler") == 0) {

		printf("Euler in time\n");
		printf("dt: %f\n", dt);

		/* Temporal array for previous time step */
		double *d_Y_tmp;
		hipMalloc(&d_Y_tmp, size * sizeof(double));

		/* GPU parallel approach */
		if (strcmp(parameters.approach, "all") == 0) {
			for (int k = 1; k <= parameters.L; k++) { 
				hipMemcpy(d_Y_tmp, d_Y, size * sizeof(double), hipMemcpyDeviceToDevice);
				RHSEuler<<<DG(size), DB>>>(parameters, DM, d_Y, d_Y_tmp, dt);
			}
		} else if (strcmp(parameters.approach, "block") == 0) {
			for (int k = 1; k <= parameters.L; k++) { 
				hipMemcpy(d_Y_tmp, d_Y, size * sizeof(double), hipMemcpyDeviceToDevice);
				RHSEulerBlock<<<n_sim, DB>>>(parameters, DM, d_Y, d_Y_tmp, dt);
			}
		}

		hipFree(d_Y_tmp);

	} else if (strcmp(parameters.time, "RK4") == 0) {

		printf("RK4 in time \n");
		printf("dt: %f\n", dt);

		/* Temporal arrays for previous ks */
		double *d_Y_tmp, *d_k1, *d_k2, *d_k3, *d_k4, *d_ktmp;

		hipMalloc(&d_k1, size * sizeof(double));
		hipMalloc(&d_k2, size * sizeof(double));
		hipMalloc(&d_k3, size * sizeof(double));
		hipMalloc(&d_k4, size * sizeof(double));
		hipMalloc(&d_ktmp, size * sizeof(double));
		hipMalloc(&d_Y_tmp, size * sizeof(double));
		hipMemset(d_k1, 0, size * sizeof(double));
		hipMemset(d_k2, 0, size * sizeof(double));
		hipMemset(d_k3, 0, size * sizeof(double));
		hipMemset(d_k4, 0, size * sizeof(double));
		hipMemset(d_ktmp, 0, size * sizeof(double));

		/* GPU parallel approach */
		if (strcmp(parameters.approach, "all") == 0) {
			for (int k = 1; k <= parameters.L; k++) { 
				hipMemcpy(d_Y_tmp, d_Y, size * sizeof(double), hipMemcpyDeviceToDevice); // Y_{t-1}
				RHSvec<<<DG(size), DB>>>(parameters, DM, d_k1, d_Y_tmp); // Compute k1
				sumVector<<<DG(size), DB>>>(parameters, d_ktmp, d_Y_tmp, d_k1, 0.5*dt, size); // Y_{t-1} + 0.5*dt*k1
				RHSvec<<<DG(size), DB>>>(parameters, DM, d_k2, d_ktmp); // Compute k2
				sumVector<<<DG(size), DB>>>(parameters, d_ktmp, d_Y_tmp, d_k2, 0.5 * dt, size); // Y_{t-1} + 0.5*dt*k2
				RHSvec<<<DG(size), DB>>>(parameters, DM, d_k3, d_ktmp); // Compute k3
				sumVector<<<DG(size), DB>>>(parameters, d_ktmp, d_Y_tmp, d_k3, dt, size); // Y_{t-1} + dt*k3
				RHSvec<<<DG(size), DB>>>(parameters, DM, d_k4, d_ktmp); // Compute k4
				RHSRK4<<<DG(size), DB>>>(parameters, DM, d_Y, d_Y_tmp, d_k1, d_k2, d_k3, d_k4, dt); // RK4
			}
		} else if (strcmp(parameters.approach, "block") == 0) {
			for (int k = 1; k <= parameters.L; k++) { 
				hipMemcpy(d_Y_tmp, d_Y, size * sizeof(double), hipMemcpyDeviceToDevice); // Y_{t-1}
				RHSvecBlock<<<n_sim, DB>>>(parameters, DM, d_k1, d_Y_tmp); // Compute k1
				//RHSvec<<<DG(size), DB>>>(parameters, DM, d_k1, d_Y_tmp);
				hipDeviceSynchronize();
				sumVector<<<DG(size), DB>>>(parameters, d_ktmp, d_Y_tmp, d_k1, 0.5*dt, size); // Y_{t-1} + 0.5*dt*k1
				//hipDeviceSynchronize();
				RHSvecBlock<<<n_sim, DB>>>(parameters, DM, d_k2, d_ktmp); // Compute k2
				//RHSvec<<<DG(size), DB>>>(parameters, DM, d_k2, d_ktmp);
				hipDeviceSynchronize();
				sumVector<<<DG(size), DB>>>(parameters, d_ktmp, d_Y_tmp, d_k2, 0.5 * dt, size); // Y_{t-1} + 0.5*dt*k2
				//hipDeviceSynchronize();
				RHSvecBlock<<<n_sim, DB>>>(parameters, DM, d_k3, d_ktmp); // Compute k3
				//RHSvec<<<DG(size), DB>>>(parameters, DM, d_k3, d_ktmp);
				hipDeviceSynchronize();
				sumVector<<<DG(size), DB>>>(parameters, d_ktmp, d_Y_tmp, d_k3, dt, size); // Y_{t-1} + dt*k3
				//hipDeviceSynchronize();
				RHSvecBlock<<<n_sim, DB>>>(parameters, DM, d_k4, d_ktmp); // Compute k4
				//RHSvec<<<DG(size), DB>>>(parameters, DM, d_k4, d_ktmp);
				hipDeviceSynchronize();
				RHSRK4Block<<<n_sim, DB>>>(parameters, DM, d_Y, d_Y_tmp, d_k1, d_k2, d_k3, d_k4, dt); // RK4
				//hipDeviceSynchronize();
			}
		}

		hipFree(d_Y_tmp);
		hipFree(d_k1);
		hipFree(d_k2);
		hipFree(d_k3);
		hipFree(d_k4);
		hipFree(d_ktmp);
	}
}

void fillInitialConditions(Parameters parameters, double *d_sims, int save) {
	/* Initial wildfire focus */
	double dx_ign, dy_ign, x_ign, y_ign;

	if (parameters.x_ign_n * parameters.y_ign_n > 1) {
		dx_ign = (parameters.x_ign_max - parameters.x_ign_min) / (parameters.x_ign_n - 1);
		dy_ign = (parameters.y_ign_max - parameters.y_ign_min) / (parameters.y_ign_n - 1);	
	} else {
		dx_ign = 1;
		dy_ign = 1;
	}

	/* To save IC */
	char sim_name[40];

	/* Temporal arrays */
	double *d_tmp;
	double *h_tmp = (double *) malloc(parameters.M * parameters.N * sizeof(double));

	hipMalloc(&d_tmp, parameters.M * parameters.N * sizeof(double));
	hipMemset(d_tmp, 0, parameters.M * parameters.N  * sizeof(double));
	
	/* Fill initial conditions according to ignitions points */
	int sim_ = 0;
	for (int i=0; i < parameters.y_ign_n; i++) {
		for (int j=0; j < parameters.x_ign_n; j++) {		

			/* Coordinates of ignition point */
			x_ign = parameters.x_ign_min + dx_ign * j;	
			y_ign = parameters.y_ign_min + dy_ign * i;

			/* Compute initial condition for temperature */
			U0<<<DG(parameters.M * parameters.N), DB>>>(parameters, d_tmp, x_ign, y_ign);
			hipMemcpy(d_sims + 2*sim_*parameters.M*parameters.N, 
				d_tmp, parameters.M * parameters.N * sizeof(double), hipMemcpyDeviceToDevice);
			
			/* Save temperature IC */
			if (save) {
				hipMemcpy(h_tmp, d_sims + 2*sim_*parameters.M*parameters.N, 
					parameters.M * parameters.N * sizeof(double), hipMemcpyDeviceToHost);
				memset(&sim_name[0], 0, sizeof(sim_name)); // Reset simulation name
				sprintf(sim_name, "%s/%s0_%d%d.txt", parameters.dir, "U", i, j); // Simulation name
				//sprintf(sim_name, "test/output/%s0_%d%d.txt", "U", i, j); // Simulation name
				saveApproximation(sim_name, h_tmp, parameters.M, parameters.N); // Save U0
			}
			
			/* Compute initial condition for fuel */
			B0<<<DG(parameters.M * parameters.N), DB>>>(parameters, d_tmp);
			hipMemcpy(d_sims + (2*sim_+1) * parameters.M * parameters.N, 
				d_tmp, parameters.M * parameters.N * sizeof(double), hipMemcpyDeviceToDevice);
			
			/* Save fuel IC */
			if (save) {
				hipMemcpy(h_tmp, d_sims + (2*sim_+1) * parameters.M * parameters.N, parameters.M * parameters.N * sizeof(double), hipMemcpyDeviceToHost);
				memset(&sim_name[0], 0, sizeof(sim_name)); // Reset simulation name
				sprintf(sim_name, "%s/%s0_%d%d.txt", parameters.dir, "B", i, j); // Simulation name
				//sprintf(sim_name, "test/output/%s0_%d%d.txt", "B", i, j); // Simulation name
				saveApproximation(sim_name, h_tmp, parameters.M, parameters.N);	// Save B0	 
			}		

			sim_++;		
		}		
	}

	/* Free memory */
	hipFree(d_tmp);
	free(h_tmp);
}

void saveResults(Parameters parameters, double *h_sims) {
	/* Simulation name */
	char sim_name[40];

	/* Temporal array */
	double *h_tmp = (double *) malloc(parameters.M * parameters.N * sizeof(double));

	int sim_ = 0;
	for (int i=0; i < parameters.y_ign_n; i++) {
		for (int j=0; j < parameters.x_ign_n; j++) {	

			/* Temperature */
			memcpy(h_tmp, h_sims + 2*sim_*parameters.M*parameters.N, 
				parameters.M * parameters.N * sizeof(double));
			memset(&sim_name[0], 0, sizeof(sim_name)); // Reset simulation name
			sprintf(sim_name, "%s/%s_%d%d.txt", parameters.dir, "U", i, j); // Simulation name
			//sprintf(sim_name, "test/output/%s_%d%d.txt", "U", i, j); // Simulation name
			saveApproximation(sim_name, h_tmp, parameters.M, parameters.N); // Save U

			/* Fuel */
			memcpy(h_tmp, h_sims + (2*sim_ + 1)*parameters.M*parameters.N, 
				parameters.M * parameters.N * sizeof(double));
			memset(&sim_name[0], 0, sizeof(sim_name)); // Reset simulation name
			sprintf(sim_name, "%s/%s_%d%d.txt", parameters.dir,"B", i, j); // Simulation name
			//sprintf(sim_name, "test/output/%s_%d%d.txt", "B", i, j); // Simulation name
			saveApproximation(sim_name, h_tmp, parameters.M, parameters.N);	// Save B	

			sim_++;
		}
	}
}

void wildfire(Parameters parameters) {

	/* Log file with parameters info */
	char log_name[100];
	sprintf(log_name, "%s/log.txt", parameters.dir);
	FILE *log = fopen(log_name, "w");

	/* Kernel Parameters */
	int n_sim = parameters.x_ign_n * parameters.y_ign_n; // Number of wildfire simulations
	int size = 2 * n_sim * parameters.M * parameters.N;

	/* Domain differentials */
	double dx = (parameters.x_max - parameters.x_min) / (parameters.N-1);
	double dy = (parameters.y_max - parameters.y_min) / (parameters.M-1);
	double dt = parameters.t_max / parameters.L;

	/* Memory for simulations */
	double *h_sims = (double *) malloc(size * sizeof(double));
	double *d_sims;	
	
	/* Domain vectors */
	double *h_x = (double *) malloc(parameters.N * sizeof(double));
	double *h_y = (double *) malloc(parameters.M * sizeof(double));
	double *d_x, *d_y;

	/* Write parameters in log */
	fprintf(log, "Simulation ID: %s\n", parameters.sim_id);
	fprintf(log, "Number of numerical simulations: %d\n", parameters.x_ign_n * parameters.y_ign_n);
	fprintf(log, "Parallel approach: %s\n", parameters.approach);
	fprintf(log, "\nIgnition points\n");
	fprintf(log, "----------------\n");
	fprintf(log, "%d in x, %d in y\n", parameters.x_ign_n, parameters.y_ign_n);
	fprintf(log, "Domain: [%f, %f]x[%f, %f]\n", parameters.x_ign_min, parameters.x_ign_max,
	parameters.y_ign_min, parameters.y_ign_max);
	fprintf(log, "\nSpace\n");
	fprintf(log, "------\n");	
	fprintf(log, "Domain: [%f, %f]x[%f, %f]\n", parameters.x_min, parameters.x_max, 
		parameters.y_min, parameters.y_max);
	fprintf(log, "Method: %s\n", parameters.spatial);
	fprintf(log, "M: %d\n", parameters.M);
	fprintf(log, "N: %d\n", parameters.N);
	fprintf(log, "dx: %f\n", dx);
	fprintf(log, "dy: %f\n", dy);
	fprintf(log, "\nTime\n");
	fprintf(log, "------\n");	
	fprintf(log, "Domain: [0, %f]\n", parameters.t_max);
	fprintf(log, "Method: %s\n", parameters.time);
	fprintf(log, "L: %d\n", parameters.L);
	fprintf(log, "dt: %f\n", dt);		
	fclose(log);

	/* Differentiation Matrices */
	
	// Struct for matrices
	DiffMats DM; 

	// Host arrays
	double *h_Dx = (double *) malloc(parameters.N * parameters.N * sizeof(double));
	double *h_Dxx = (double *) malloc(parameters.N * parameters.N * sizeof(double));
	double *h_Dy = (double *) malloc(parameters.M * parameters.M * sizeof(double));
	double *h_Dyy = (double *) malloc(parameters.M * parameters.M * sizeof(double));

	// Device arrays
	double *d_Dx, *d_Dy, *d_Dxx, *d_Dyy;

	/* Device memory allocation */
	hipMalloc(&d_sims, size * sizeof(double));
	hipMalloc(&d_x, parameters.N * sizeof(double));
	hipMalloc(&d_y, parameters.M * sizeof(double));
	hipMalloc(&d_Dx, parameters.N * parameters.N * sizeof(double));
	hipMalloc(&d_Dy, parameters.M * parameters.M * sizeof(double));
	hipMalloc(&d_Dxx, parameters.N * parameters.N * sizeof(double));
	hipMalloc(&d_Dyy, parameters.M * parameters.M * sizeof(double));

	/* Copy from host to device */
	hipMemcpy(d_sims, h_sims, size * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_x, h_x, parameters.N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_y, h_y, parameters.M * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_Dx, h_Dx, parameters.N * parameters.N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_Dy, h_Dy, parameters.M * parameters.M * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_Dxx, h_Dxx, parameters.N * parameters.N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_Dyy, h_Dyy, parameters.M * parameters.M * sizeof(double), hipMemcpyHostToDevice);

	/* Fill spatial domain and differentiation matrices */
	if (strcmp(parameters.spatial, "FD") == 0) {		
		// Spatial domain
		printf("Finite Difference in space\n");
		printf("dx: %f\n", dx);
		printf("dy: %f\n", dy);	

		// fprintf(log, "Finite Difference in space\n");
		// fprintf(log, "dx: %f\n", dx);
		// fprintf(log, "dy: %f\n", dy);	

		fillVectorKernel<<<DG(parameters.M * parameters.N), DB>>>(d_x, dx, parameters.N);
		fillVectorKernel<<<DG(parameters.M * parameters.N), DB>>>(d_y, dy, parameters.M);

		// Differentiation matrices
		FD1Kernel<<<DG(parameters.M * parameters.N), DB>>>(d_Dx, parameters.N, dx);
		FD1Kernel<<<DG(parameters.M * parameters.N), DB>>>(d_Dy, parameters.M, dy);
		FD2Kernel<<<DG(parameters.M * parameters.N), DB>>>(d_Dxx, parameters.N, dx);
		FD2Kernel<<<DG(parameters.M * parameters.N), DB>>>(d_Dyy, parameters.M, dy);

	} else if (strcmp(parameters.spatial, "Cheb") == 0) {
		// Spatial domain
		printf("Chebyshev in space\n");
		//fprintf(log, "Chebyshev in space\n");

		ChebyshevNodes<<<DG(parameters.M * parameters.N), DB>>>(d_x, parameters.N - 1);
		ChebyshevNodes<<<DG(parameters.M * parameters.N), DB>>>(d_y, parameters.M - 1);

		// Differentiation matrices
		ChebyshevMatrix<<<DG(parameters.M * parameters.N), DB>>>(d_Dx, d_x, parameters.N - 1);
		ChebyshevMatrix<<<DG(parameters.M * parameters.N), DB>>>(d_Dy, d_y, parameters.M - 1);
		Chebyshev2Matrix<<<DG(parameters.M * parameters.N), DB>>>(d_Dxx, d_Dx, parameters.N - 1);
		Chebyshev2Matrix<<<DG(parameters.M * parameters.N), DB>>>(d_Dyy, d_Dy, parameters.M - 1);

	} else {
		printf("Spatial domain error\n");
		exit(0);
	}	
	
	/* Copy spatial domain to constant memory */
	hipMemcpyToSymbol(HIP_SYMBOL(buffer), d_x, parameters.N * sizeof(double), 0, hipMemcpyDeviceToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(buffer), d_y, parameters.M * sizeof(double), parameters.N * sizeof(double), hipMemcpyDeviceToDevice);
	
	DM.Dx = d_Dx;
	DM.Dy = d_Dy;
	DM.Dxx = d_Dxx;
	DM.Dyy = d_Dyy;

	/* Fill initial conditions */	
	fillInitialConditions(parameters, d_sims, 1);

	/* ODE Integration */
	ODESolver(parameters, DM, d_sims, dt);
	// double *d_tmp;
	// hipMalloc(&d_tmp, 2 * parameters.M * parameters.N * sizeof(double));
	// hipMemset(d_tmp, 0, 2 * parameters.M * parameters.N  * sizeof(double));

	// simulationBlock<<<n_sim, DB, DB * sizeof(double)>>>(parameters, DM, d_sims, d_tmp, dt);

	//hipDeviceSynchronize();

	/* Copy approximations to host */
	hipMemcpy(h_sims, d_sims, size * sizeof(double), hipMemcpyDeviceToHost);

	/* Save */
	saveResults(parameters, h_sims);

	/* Memory free */
	hipFree(d_sims);
	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_Dx);
	hipFree(d_Dy);
	hipFree(d_Dxx);
	hipFree(d_Dyy);
	free(h_sims);
	free(h_x);
	free(h_y);
	free(h_Dx);
	free(h_Dy);
	free(h_Dxx);
	free(h_Dyy);
}