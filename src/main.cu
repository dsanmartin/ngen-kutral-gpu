#include "hip/hip_runtime.h"
// Low level matrix multiplication on GPU using CUDA with CURAND and CUBLAS
// C(m,n) = A(m,k) * B(k,n)
#include <stdio.h>
#include <cstdlib>
#include <stdlib.h>
#include <ctime>
#include <hipblas.h>
#include <hiprand.h>
#include "c/include/files.h"

// Fill the array A(nr_rows_A, nr_cols_A) with random numbers on GPU
void GPU_fill_rand(float *A, int nr_rows_A, int nr_cols_A) {
	// Create a pseudo-random number generator
	hiprandGenerator_t prng;
	hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

	// Set the seed for the random number generator using the system clock
	hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());

	// Fill the array with random numbers on the device
	hiprandGenerateUniform(prng, A, nr_rows_A * nr_cols_A);
}

// Multiply the arrays A and B on GPU and save the result in C
// C(m,n) = A(m,k) * B(k,n)
void gpu_blas_mmul(const float *A, const float *B, float *C, const int m, const int k, const int n) {
	int lda=m,ldb=k,ldc=m;
	const float alf = 1;
	const float bet = 0;
	const float *alpha = &alf;
	const float *beta = &bet;

	// Create a handle for CUBLAS
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	// Do the actual multiplication
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);

	// Destroy the handle
	hipblasDestroy(handle);
}


//Print matrix A(nr_rows_A, nr_cols_A) storage in column-major format
void printMatrix(const double *A, int nr_rows_A, int nr_cols_A) {
    for(int i = 0; i < nr_rows_A; ++i){
        for(int j = 0; j < nr_cols_A; ++j){
            printf("%lf ", A[j * nr_rows_A + i]);	
        }
        printf("\n");
    }
    printf("\n");
}

int main() {
	// Allocate 3 arrays on CPU
	int nr_rows_A, nr_cols_A, nr_rows_B, nr_cols_B, nr_rows_C, nr_cols_C;

	// for simplicity we are going to use square arrays
	nr_rows_A = 4;
	nr_cols_A = 2;
	nr_rows_B = 2;
	nr_cols_B = 3;
	nr_rows_C = 3;
	nr_cols_C = 3;
	
	double *h_A = (double *)malloc(nr_rows_A * nr_cols_A * sizeof(double));
	float *h_B = (float *)malloc(nr_rows_B * nr_cols_B * sizeof(float));
	float *h_C = (float *)malloc(nr_rows_C * nr_cols_C * sizeof(float));

	hola();
	//readInput(h_A, nr_rows_A, nr_cols_A);

	// // Allocate 3 arrays on GPU
	// float *d_A, *d_B, *d_C;
	// hipMalloc(&d_A,nr_rows_A * nr_cols_A * sizeof(float));
	// hipMalloc(&d_B,nr_rows_B * nr_cols_B * sizeof(float));
	// hipMalloc(&d_C,nr_rows_C * nr_cols_C * sizeof(float));

	// h_A[0] = 1;
	// h_A[1] = 2;
	// h_A[2] = 3;
	// h_A[3] = 4;
	// h_A[4] = 5;
	// h_A[5] = 6;
	// h_B[0] = 1;
	// h_B[1] = 2;
	// h_B[2] = 3;
	// h_B[3] = 4;
	// h_B[4] = 5;
	// h_B[5] = 6;


	// // If you already have useful values in A and B you can copy them in GPU:
	// hipMemcpy(d_A, h_A, nr_rows_A * nr_cols_A * sizeof(float), hipMemcpyHostToDevice);
	// hipMemcpy(d_B, h_B, nr_rows_B * nr_cols_B * sizeof(float), hipMemcpyHostToDevice);

	// // Fill the arrays A and B on GPU with random numbers
	// // GPU_fill_rand(d_A, nr_rows_A, nr_cols_A);
	// // GPU_fill_rand(d_B, nr_rows_B, nr_cols_B);

	// // Optionally we can copy the data back on CPU and print the arrays
	// hipMemcpy(h_A,d_A,nr_rows_A * nr_cols_A * sizeof(float),hipMemcpyDeviceToHost);
	// hipMemcpy(h_B,d_B,nr_rows_B * nr_cols_B * sizeof(float),hipMemcpyDeviceToHost);
  printf("A=\n");
	printMatrix(h_A, nr_rows_A, nr_cols_A);
  // printf("B=\n");
	// printMatrix(h_B, nr_rows_B, nr_cols_B);

	// // Multiply A and B on GPU
	// gpu_blas_mmul(d_A, d_B, d_C, nr_rows_A, nr_cols_A, nr_cols_B);

	// // Copy (and print) the result on host memory
	// hipMemcpy(h_C,d_C,nr_rows_C * nr_cols_C * sizeof(float),hipMemcpyDeviceToHost);
  // printf("C=\n");
	// printMatrix(h_C, nr_rows_C, nr_cols_C);

	// //Free GPU memory
	// hipFree(d_A);
	// hipFree(d_B);
	// hipFree(d_C);	

	// Free CPU memory
	free(h_A);
	free(h_B);
	free(h_C);

	return 0;
}